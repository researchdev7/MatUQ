#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
#include <hiprand/hiprand_kernel.h>
extern "C" __global__


void compute_lattice_second_frame(
    const float* a_ik,
    const float* rpos_ij_e,
    const float* dist2_min_e,
    const float* tvecs_n,
    const long long int* batch_i,
    const long long int* edge_ij_e,
    const long long int N,
    const long long int H,
    const long long int E,
    const long long int K_,
    const double dist_max,
    const double wscale,
    const float* rveclens_n,
    const double cutoff_radius,
    hiprandState *state_buff,
    unsigned long long seed,
    float* dx_first,
    float* dy_first,
    float* dz_first,
    float* dx_second,
    float* dy_second,
    float* dz_second
    ){
    const long long int tid = (long long int)blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= E*H) return;

    const long long int k = tid%H;
    const long long int e = tid/H;
    const long long int i = edge_ij_e[e];
    const long long int j = edge_ij_e[E+e];

    if (i != j) return;

    const long long int n = batch_i[i];
    rpos_ij_e += e*3;
    const float r_ijx = rpos_ij_e[0];
    const float r_ijy = rpos_ij_e[1];
    const float r_ijz = rpos_ij_e[2];
    tvecs_n += n*9;
    const float t1_x = tvecs_n[0];
    const float t1_y = tvecs_n[1];
    const float t1_z = tvecs_n[2];
    const float t2_x = tvecs_n[3];
    const float t2_y = tvecs_n[4];
    const float t2_z = tvecs_n[5];
    const float t3_x = tvecs_n[6];
    const float t3_y = tvecs_n[7];
    const float t3_z = tvecs_n[8];
    const float a = a_ik[i*H + k];
    const int R = LATTICE_RANGE;
    const float Rf = (float)LATTICE_RANGE;

    rveclens_n += n*3;
    const float rvl1 = rveclens_n[0];
    const float rvl2 = rveclens_n[1];
    const float rvl3 = rveclens_n[2];

    float cutoff = (float)cutoff_radius;
    int R1 = LATTICE_RANGE, R2 = LATTICE_RANGE, R3 = LATTICE_RANGE;
    if (cutoff != 0.0f)
    {
        if (cutoff < 0) {
            // Better sync the threads in each block?
            // -> disabled due to thread stucking
            // float a_max = a;
            // for (int t = 0; t < THREAD_NUM; t++)
            //     a_max = max(a_max, a_ik[i*H + t]);
            //cutoff = sqrt(-0.5f/a_max)*(-cutoff);
            cutoff = sqrt(-0.5f/a)*(-cutoff);
        }
        R1 = ceil((cutoff + 0.01f)*rvl1/(2.0*HIP_PI_F));
        R2 = ceil((cutoff + 0.01f)*rvl2/(2.0*HIP_PI_F));
        R3 = ceil((cutoff + 0.01f)*rvl3/(2.0*HIP_PI_F));

        #if MINIMUM_RANGE > 0
        R1 = max(R1, MINIMUM_RANGE);
        R2 = max(R2, MINIMUM_RANGE);
        R3 = max(R3, MINIMUM_RANGE);
        #endif
    }

    float d2_min = 1e10;
    long long int idx = 1000*tid;
    long long int max_idx = E*H*1000;
    float sum = 0;
    hiprandState *state = state_buff + tid;
    hiprand_init(seed, tid, 0, state);

    for (float n1 = -R1; n1 <= R1; n1++)
    for (float n2 = -R2; n2 <= R2; n2++)
    for (float n3 = -R3; n3 <= R3; n3++)
    {
        float dx = r_ijx + t1_x*n1 + t2_x*n2 + t3_x*n3;
        float dy = r_ijy + t1_y*n1 + t2_y*n2 + t3_y*n3;
        float dz = r_ijz + t1_z*n1 + t2_z*n2 + t3_z*n3;
        float d2 = dx*dx + dy*dy + dz*dz;
        float d = sqrtf(d2);

        float rand = (hiprand_uniform(state)-0.5)*SYMM_BREAK_NOISE;
        
        if (d2_min > d2 + rand && d2 > 1e-5)
        {
        float dx_norm = dx/d;
        float dy_norm = dy/d;
        float dz_norm = dz/d;

        float cos_term = dx_norm * dx_first[i*H+k] + dy_norm * dy_first[i*H+k] + dz_norm * dz_first[i*H+k];

        if (1 - fabsf(cos_term) > 1e-5){
            if (cos_term<0){
            dx_second[i*H+k] = -dx_norm;
            dy_second[i*H+k] = -dy_norm;
            dz_second[i*H+k] = -dz_norm;
            } else {
            dx_second[i*H+k] = dx_norm;
            dy_second[i*H+k] = dy_norm;
            dz_second[i*H+k] = dz_norm;
            }
            d2_min = d2 + rand;
        }
        }
        idx += 1;
    }
}