#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
#include <hiprand/hiprand_kernel.h>
extern "C" __global__


void real_enc_sigma_fwd(
    const float* a_ik,
    const float* rpos_ij_e,
    const float* dist2_min_e,
    const float* tvecs_n,
    const long long int* batch_i,
    const long long int* edge_ij_e,
    const long long int N,
    const long long int H,
    const long long int E,
    const long long int K_,
    const double dist_max,
    const double wscale,
    const long long int W_num,
    const float* rveclens_n,
    const double cutoff_radius,
    float* sigma_mat,
    hiprandState *state_buff,
    unsigned long long seed,
    int* edge_ij_e_select,
    float* sigma_mat_max){
    const long long int tid = (long long int)blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= E*H) return;

    const long long int k = tid%H;
    const long long int e = tid/H;
    const long long int i = edge_ij_e[e];
    const long long int j = edge_ij_e[E+e];
    const long long int n = batch_i[i];
    rpos_ij_e += e*3;
    const float r_ijx = rpos_ij_e[0];
    const float r_ijy = rpos_ij_e[1];
    const float r_ijz = rpos_ij_e[2];
    tvecs_n += n*9;
    const float t1_x = tvecs_n[0];
    const float t1_y = tvecs_n[1];
    const float t1_z = tvecs_n[2];
    const float t2_x = tvecs_n[3];
    const float t2_y = tvecs_n[4];
    const float t2_z = tvecs_n[5];
    const float t3_x = tvecs_n[6];
    const float t3_y = tvecs_n[7];
    const float t3_z = tvecs_n[8];
    const float a = a_ik[i*H + k];
    const int R = LATTICE_RANGE;
    const float Rf = (float)LATTICE_RANGE;

    #if VPE_DIM > 0
    __shared__ float shared_v[THREAD_NUM][VPE_DIM+1];
    float *sv = shared_v[threadIdx.x];

    for (int dim = 0; dim < VPE_DIM; dim++)
        sv[dim] = 0;
    const float reci_ws_sqrt2 = 1.0f/((float)wscale*sqrt(2.0f));
    const float mu0 = (float)dist_max/VPE_DIM;
    #endif

    rveclens_n += n*3;
    const float rvl1 = rveclens_n[0];
    const float rvl2 = rveclens_n[1];
    const float rvl3 = rveclens_n[2];

    float cutoff = (float)cutoff_radius;
    int R1 = LATTICE_RANGE, R2 = LATTICE_RANGE, R3 = LATTICE_RANGE;
    if (cutoff != 0.0f)
    {
        if (cutoff < 0) {
            // Better sync the threads in each block?
            // -> disabled due to thread stucking
            // float a_max = a;
            // for (int t = 0; t < THREAD_NUM; t++)
            //     a_max = max(a_max, a_ik[i*H + t]);
            //cutoff = sqrt(-0.5f/a_max)*(-cutoff);
            cutoff = sqrt(-0.5f/a)*(-cutoff);
        }
        R1 = ceil((cutoff + 0.01f)*rvl1/(2.0*HIP_PI_F));
        R2 = ceil((cutoff + 0.01f)*rvl2/(2.0*HIP_PI_F));
        R3 = ceil((cutoff + 0.01f)*rvl3/(2.0*HIP_PI_F));

        #if MINIMUM_RANGE > 0
        R1 = max(R1, MINIMUM_RANGE);
        R2 = max(R2, MINIMUM_RANGE);
        R3 = max(R3, MINIMUM_RANGE);
        #endif
    }

    float d2min = 1e10;
    if (1 || dist2_min_e == NULL)
    {
        for (float n1 = -R1; n1 <= R1; n1++)
        for (float n2 = -R2; n2 <= R2; n2++)
        for (float n3 = -R3; n3 <= R3; n3++)
        {
            float dx = r_ijx + t1_x*n1 + t2_x*n2 + t3_x*n3;
            float dy = r_ijy + t1_y*n1 + t2_y*n2 + t3_y*n3;
            float dz = r_ijz + t1_z*n1 + t2_z*n2 + t3_z*n3;
            float d2 = dx*dx + dy*dy + dz*dz;
            // float dx = fmaf(t1_x, n1, fmaf(t2_x, n2, fmaf(t3_x, n3, r_ijx)));
            // float dy = fmaf(t1_y, n1, fmaf(t2_y, n2, fmaf(t3_y, n3, r_ijy)));
            // float dz = fmaf(t1_z, n1, fmaf(t2_z, n2, fmaf(t3_z, n3, r_ijz)));
            // float d2 = fmaf(dx,dx, fmaf(dy,dy, dz*dz));
            d2min = fminf(d2min, d2);
        }
    } else {
        d2min = dist2_min_e[e];
    }
    float sum = 0;

    float sum_wo_0 = 0;
    float d2_max = 100000000;
    int atom = edge_ij_e[e];
    long long int idx = 1000 * tid;
    long long int divval = E * H * 1000;

    hiprandState *state = state_buff + tid;
    hiprand_init(seed, tid, 0, state);

    for (float n1 = -R1; n1 <= R1; n1++)
    for (float n2 = -R2; n2 <= R2; n2++)
    for (float n3 = -R3; n3 <= R3; n3++)
    {
        float dx = r_ijx + t1_x*n1 + t2_x*n2 + t3_x*n3;
        float dy = r_ijy + t1_y*n1 + t2_y*n2 + t3_y*n3;
        float dz = r_ijz + t1_z*n1 + t2_z*n2 + t3_z*n3;
        float d2 = dx*dx + dy*dy + dz*dz;
        // float dx = fmaf(t1_x, n1, fmaf(t2_x, n2, fmaf(t3_x, n3, r_ijx)));
        // float dy = fmaf(t1_y, n1, fmaf(t2_y, n2, fmaf(t3_y, n3, r_ijy)));
        // float dz = fmaf(t1_z, n1, fmaf(t2_z, n2, fmaf(t3_z, n3, r_ijz)));
        // float d2 = fmaf(dx,dx, fmaf(dy,dy, dz*dz));
        float w = expf(a*(d2 - d2min));

        sum += w;

        sigma_mat[tid * 6 + 0] += dx * dx * w;
        sigma_mat[tid * 6 + 1] += dx * dy * w;
        sigma_mat[tid * 6 + 2] += dx * dz * w;
        sigma_mat[tid * 6 + 3] += dy * dy * w;
        sigma_mat[tid * 6 + 4] += dy * dz * w;
        sigma_mat[tid * 6 + 5] += dz * dz * w;

        long long int div_result = idx % divval;

        float rand = (hiprand_uniform(state)-0.5)*SYMM_BREAK_NOISE;

        if (d2 > 1e-5 && d2_max > d2 + rand)
        {
        d2_max = d2+rand;
        sigma_mat_max[(atom*H+k) * 6 + 0] = dx * dx / d2;
        sigma_mat_max[(atom*H+k) * 6 + 1] = dx * dy / d2;
        sigma_mat_max[(atom*H+k) * 6 + 2] = dx * dz / d2;
        sigma_mat_max[(atom*H+k) * 6 + 3] = dy * dy / d2;
        sigma_mat_max[(atom*H+k) * 6 + 4] = dy * dz / d2;
        sigma_mat_max[(atom*H+k) * 6 + 5] = dz * dz / d2;
        }
        idx += 1;
    }

    for (int dim = 0; dim < 6; dim++){
        sigma_mat[tid * 6 + dim] = sigma_mat[tid * 6 + dim] / sum;
    }
}

